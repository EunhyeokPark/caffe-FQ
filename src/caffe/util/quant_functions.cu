#include "hip/hip_runtime.h"
#include <cmath>
#include <cfloat>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

#include "caffe/common.hpp"
#include "caffe/util/quant_functions.hpp"

namespace caffe
{
  
// find level difference for applying quantization
template <typename Dtype>
QuantOpt<Dtype> find_quant_opt(const Dtype* input, const int N, const int num_level)
{
  thrust::device_ptr<const Dtype> t_ptr = thrust::device_pointer_cast<const Dtype>(input);
  thrust::pair<thrust::device_ptr<const Dtype>, thrust::device_ptr<const Dtype> > rtn = 
    thrust::minmax_element(t_ptr, t_ptr+N);

  const Dtype min_val = *(rtn.first);
  const Dtype max_val = *(rtn.second);

  // positive only quantization, e.g. after ReLU
  if(min_val > -1 * FLT_EPSILON){
    Dtype diff =  powf(2., ceil( log2(max_val/(num_level-1)) ));
    return QuantOpt<Dtype>(diff, 0, num_level - 1);
  }
  else{// positive-negative all range quantization
    // if num_level is odd, then both sides are symmetry
    // otherwise, negative side has one more level
    int pos_level = (num_level-1) / 2;
    int neg_level = -1 * num_level / 2;

    Dtype diff = max(min_val / neg_level, max_val / pos_level);
    diff = powf(2., ceil(log2(diff)));
    return QuantOpt<Dtype>(diff, neg_level, pos_level);
  }
  // other cases???
}

template
QuantOpt<float> find_quant_opt(const float* input, const int N, const int num_level);

template 
QuantOpt<double> find_quant_opt(const double* input, const int N, const int num_level);

template <typename Dtype>
__global__ void quantize_kernel(const int n, 
  const Dtype* input, Dtype* output, const QuantOpt<Dtype> opt)
{
  CUDA_KERNEL_LOOP(i, n)
  {
    Dtype rounded = round(input[i] / opt.diff);
    rounded = (rounded > opt.max_level) ? opt.max_level :
      (rounded < opt.min_level ? opt.min_level : rounded);
    output[i] = rounded * opt.diff;
  }
}

// apply quantization for input data with N elements.
template <typename Dtype>
void quantize(const Dtype* input, Dtype* output, const int N, const QuantOpt<Dtype>& opt)
{
  quantize_kernel<Dtype><<< CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS >>>
    (N, input, output, opt);
}

template
void quantize(const float* input, float* output, const int N, const QuantOpt<float>& opt);

template
void quantize(const double* input, double* output, const int N, const QuantOpt<double>& opt);

} // namespace caffe