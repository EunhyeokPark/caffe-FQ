#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/fq_inner_product_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/quant_functions.hpp"

namespace caffe {

template <typename Dtype>
void FQInnerProductLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* weight = this->blobs_[0]->gpu_data();
  
  Dtype* quant = this->quant_.mutable_gpu_data();
  Dtype* quant_data = this->blobs_[this->blobs_.size()-1]->mutable_cpu_data();
  const int num_level = static_cast<int>(quant_data[0]);
  Dtype& diff = quant_data[1];
  const int min_level = static_cast<int>(quant_data[2]);
  const int max_level = static_cast<int>(quant_data[3]);

  QuantOpt<Dtype> opt = QuantOpt<Dtype>(diff, min_level, max_level);
  
  if(this->phase_ == TRAIN || diff == 0 || min_level == max_level)
  {
    opt = find_quant_opt<Dtype>(weight, this->blobs_[0]->count(), num_level);
    diff = max(diff, opt.diff);
    quant_data[2] = static_cast<Dtype>(opt.min_level);
    quant_data[3] = static_cast<Dtype>(opt.max_level);
  }  

  quantize<Dtype>(weight, quant, this->blobs_[0]->count(), opt);
  CUDA_POST_KERNEL_CHECK;

  if (this->M_ == 1) {
    caffe_gpu_gemv<Dtype>(CblasNoTrans, this->N_, this->K_, (Dtype)1.,
                         quant, bottom_data, (Dtype)0., top_data);
    if (this->bias_term_)
      caffe_gpu_axpy<Dtype>(this->N_, this->bias_multiplier_.cpu_data()[0],
                            this->blobs_[1]->gpu_data(), top_data);
  } else {
    caffe_gpu_gemm<Dtype>(CblasNoTrans,
                          this->transpose_ ? CblasNoTrans : CblasTrans,
                          this->M_, this->N_, this->K_, (Dtype)1.,
                          bottom_data, quant, (Dtype)0., top_data);
    if (this->bias_term_)
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, this->M_, this->N_, 1, (Dtype)1.,
                            this->bias_multiplier_.gpu_data(),
                            this->blobs_[1]->gpu_data(), (Dtype)1., top_data);
  }
}

template <typename Dtype>
void FQInnerProductLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (this->param_propagate_down_[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();

    // Gradient with respect to weight
    if (this->transpose_) {
      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
          this->K_, this->N_, this->M_,
          (Dtype)1., bottom_data, top_diff,
          (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
    } else {
      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
          this->N_, this->K_, this->M_,
          (Dtype)1., top_diff, bottom_data,
          (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
    }
  }
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    // Gradient with respect to bias
    caffe_gpu_gemv<Dtype>(CblasTrans, this->M_, this->N_, (Dtype)1., top_diff,
        this->bias_multiplier_.gpu_data(), (Dtype)1.,
        this->blobs_[1]->mutable_gpu_diff());
  }
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    // Gradient with respect to bottom data
    if (this->transpose_) {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans,
          this->M_, this->K_, this->N_,
          (Dtype)1., top_diff, this->quant_.gpu_data(),
          (Dtype)0., bottom[0]->mutable_gpu_diff());
    } else {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans,
          this->M_, this->K_, this->N_,
         (Dtype)1., top_diff, this->quant_.gpu_data(),
         (Dtype)0., bottom[0]->mutable_gpu_diff());
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(FQInnerProductLayer);

}  // namespace caffe
