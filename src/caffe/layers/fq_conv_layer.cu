#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/fq_conv_layer.hpp"
#include "caffe/util/quant_functions.hpp"

namespace caffe {

template <typename Dtype>
void FQConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  
  Dtype* quant = this->quant_.mutable_gpu_data();
  Dtype* quant_data = this->blobs_[this->blobs_.size()-1]->mutable_cpu_data();
  const int num_level = static_cast<int>(quant_data[0]);
  Dtype& diff = quant_data[1];
  const int min_level = static_cast<int>(quant_data[2]);
  const int max_level = static_cast<int>(quant_data[3]);

  QuantOpt<Dtype> opt = QuantOpt<Dtype>(diff, min_level, max_level);
  
  if(this->phase_ == TRAIN || diff == 0 || min_level == max_level)
  {
    opt = find_quant_opt<Dtype>(weight, this->blobs_[0]->count(), num_level);
    diff = max(diff, opt.diff);
    quant_data[2] = static_cast<Dtype>(opt.min_level);
    quant_data[3] = static_cast<Dtype>(opt.max_level);
  }  

  quantize<Dtype>(weight, quant, this->blobs_[0]->count(), opt);
  CUDA_POST_KERNEL_CHECK;

  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, quant,
          top_data + n * this->top_dim_);
      if (this->bias_term_) {
        const Dtype* bias = this->blobs_[1]->gpu_data();
        this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
      }
    }
  }
}

template <typename Dtype>
void FQConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* quant = this->quant_.gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, quant,
              bottom_diff + n * this->bottom_dim_);
        }
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(FQConvolutionLayer);

}  // namespace caffe
