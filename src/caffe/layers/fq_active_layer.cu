#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include "caffe/layers/fq_active_layer.hpp"
#include "caffe/util/quant_functions.hpp"

namespace caffe {

template <typename Dtype>
void FQActiveLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();

  Dtype* quant_data = this->blobs_[0]->mutable_cpu_data();
  const int num_level = static_cast<int>(quant_data[0]);
  Dtype& diff = quant_data[1];
  const int min_level = static_cast<int>(quant_data[2]);
  const int max_level = static_cast<int>(quant_data[3]);

  QuantOpt<Dtype> opt = QuantOpt<Dtype>(diff, min_level, max_level);
  
  if(this->phase_ == TRAIN || diff == 0 || min_level == max_level)
  {
    opt = find_quant_opt<Dtype>(bottom_data, count, num_level);
    diff = max(diff, opt.diff);
    quant_data[2] = static_cast<Dtype>(opt.min_level);
    quant_data[3] = static_cast<Dtype>(opt.max_level);
  }  

  quantize<Dtype>(bottom_data, top_data, count, opt);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void FQActiveLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    caffe_copy<Dtype>(count, top_diff, bottom_diff);    
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(FQActiveLayer);

}  // namespace caffe
